#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void mykernel(double* X_dev,double* Y_dev,double* temp,double* temp_vect,double* m_vect,int size,int dim,int pitch) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	int z,a;
	double m_norm=e+1,s1,s2;
		while (m_norm>e){
			m_norm=0;
			s1=0;
			s2=0;
			for (j=0;j<dim;j++){
				temp_vect[j]=0;
			}
			double dist=0;
			
			for (j=0;j<size;j++){
				for (a=0;a<dim;a++){
					dist+=pow(Y_dev[i][a]-X_dev[j][a],2);
				}
				dist=sqrt(dist);
				if (dist<=pow(s,2)){
					s1=exp(-1*pow(dist,2)/(2*pow(s,2)));
					for (z=0;z<dim;z++){
						temp[j][z]=X_dev[j][z];
						temp[j][z]*=s1;
						temp_vect[z]+=temp[j][z];
					}
					s2+=exp(-1*pow(dist,2)/(2*pow(s,2)));
				}
			}
			for(z=0;z<dim;z++){
				temp_vect[z]=temp_vect[z]/s2;
			}
			for (j=0;j<dim;j++){
				m_vect[j]=temp_vect[j]-Y_dev[i][j];
				Y_dev[i][j]=temp_vect[j];
				m_norm+=pow(m_vect[j],2)
			}
			m_norm=sqrt(m_norm);
		}
}

#define K 1
#define dimension 2
double **Create2DarrayDouble(int rows, int clmn);

int main(int argc,char **argv) {
	FILE *file;
	long size;
	double *buffer,*temp_vect,*m_vect,*X_dev,*Y_dev;
	int i,j;
	file=fopen("data.bin","rb");
	if (!file){
		printf("Unable to open file.");
		return 1;
	}
	fseek(file,0,SEEK_END);
	size=ftell(file);
	rewind(file);
	printf("%ld",size);
	buffer=(double*)malloc(sizeof(double)*(size/8));
	fread(buffer,sizeof(double),size,file);
	fclose(file);
	size=size/(8*dim);
	double **X=Create2DarrayDouble(size,dim);
	double **Y=Create2DarrayDouble(size,dim);
	double **temp=Create2DarrayDouble(size,dim);
	hipMalloc(&temp_vect,dim*sizeof(double));
	hipMalloc(&m_vect,dim*sizeof(double));
	for (i=0;i<size;i++){
		for (j=0;j<dim;j++){
			X[i,j]=buffer[i*dim+j];
			Y[i,j]=buffer[i*dim+j];
		}
	}
	size_t pitch;
	hipMallocPitch((void**)&X_dev,&pitch,dim,size);
	hipMallocPitch((void**)&Y_dev,&pitch,dim,size);
	hipMallocPitch((void**)&temp,&pitch,dim,size);
	hipMemcpy2D(X_dev,pitch,X,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyHostToDevice);
	hipMemcpy2D(Y_dev,pitch,Y,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyHostToDevice);
	int blocks=size/100;
	mykernel<<<blocks,100>>>(X_dev,Y_dev,temp,temp_vect,m_vect,size,dim,pitch);
	hipMemcpy2D(Y,pitch,Y_dev,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyDeviceToHost);
}


double **Create2DarrayDouble(int rows, int clmn){
	int i;
	double *data=(double*)malloc(rows*clmn*sizeof(double));
	double **array=(double**)malloc(rows*sizeof(double*));
	 for (int i=0; i<rows; i++)
        array[i] = &(data[clmn*i]);
    return array;
}
