#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void mykernel(double* X_dev,double* Y_dev,double* temp,double* temp_vect,double* m_vect,int size,int dim) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	double m_norm=e+1,s1,s2;
		while (m_norm>e){
			m_norm=0;
			s1=0;
			s2=0;
			for (j=0;j<dim;j++){
				temp_vect[j]=0;
			}
			for (j=0;j<N;j++){
				dist=distance(X_dev,Y_dev,i,j);
				if (dist<=pow(s,2)){
					s1_temp=exp(-1*pow(dist,2)/(2*pow(s,2)));
					for (z=0;z<dim;z++){
						temp[j][z]=X[j][z];
						temp[j][z]*=s1_temp;
						temp_vect[z]+=temp[j][z];
					}
					s2+=exp(-1*pow(dist,2)/(2*pow(s,2)));
				}
			}
			for(z=0;z<dim;z++){
				temp_vect[z]=temp_vect[z]/s2;
			}
			for (j=0;j<dim;j++){
				m_vect[j]=temp_vect[j]-Y[i][j];
				Y_dev[i][j]=temp_vect[j];
				m_norm+=pow(m_vect[j],2)
			}
			m_norm=sqrt(m_norm);
		}
}

#define K 1
#define dimension 2

int main(int argc,char **argv) {
	FILE *file;
	long size;
	double *buffer,*temp_vect,*m_vect,s1,s2,sq_temp,dist,m_norm,*X_dev,*Y_dev;
	int i,j,z,dim=dimension;
	file=fopen("data.bin","rb");
	if (!file){
		printf("Unable to open file.");
		return 1;
	}
	fseek(file,0,SEEK_END);
	size=ftell(file);
	rewind(file);
	printf("%ld",size);
	buffer=(double*)malloc(sizeof(double)*(size/8));
	fread(buffer,sizeof(double),size,file);
	fclose(file);
	size=size/(8*dim);
	double **X=Create2DarrayDouble(size,dim);
	double **Y=Create2DarrayDouble(size,dim);
	double **temp=Create2DarrayDouble(size,dim);
	hipMalloc(&temp_vect,dim*sizeof(double));
	hipMalloc(&m_vect,dim*sizeof(double));
	for (i=0;i<size;i++){
		for (j=0;j<dim;j++){
			X[i,j]=buffer[i*dim+j];
			Y[i,j]=buffer[i*dim+j];
		}
	}
	size_t pitch;
	hipMallocPitch((void**)&X_dev,&pitch,dim,size);
	hipMallocPitch((void**)&Y_dev,&pitch,dim,size);
	hipMallocPitch((void**)&temp,&pitch,dim,size);
	hipMemcpy2D(X_dev,pitch,X,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyHostToDevice);
	hipMemcpy2D(Y_dev,pitch,Y,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyHostToDevice);
	int blocks=N/100;
	mykernel<<<blocks,100>>>(X_dev,Y_dev,temp,temp_vect,m_vect,size,dim);
	hipMemcpy2D(Y,pitch,Y_dev,dim*sizeof(double),dim*sizeof(double),size,hipMemcpyDeviceToHost);
}

__device__ double distance(double** X,double** Y,int i,int j){
	int a;
	double dist=0;
	for (a=0;a<dim;a++){
		dist+=pow(Y[i][a]-X[j][a],2)
	}
	dist=sqrt(dist);
	return dist;
}

double **Create2DarrayDouble(int rows, int clmn){
	int i;
	double *data=(double*)malloc(rows*clmn*sizeof(double));
	double **array=(double**)malloc(rows*sizeof(double*));
	 for (int i=0; i<rows; i++)
        array[i] = &(data[clmn*i]);
    return array;
}
