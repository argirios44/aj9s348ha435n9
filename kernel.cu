#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
//using namespace std;

#define cols 2
const int blocksize = 128;

//Get 2 vectors/rows and multiply element-wise
__global__ void Hadamard_kernel(int* A,int* C,int* Output,int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * N;
	if (i < 128 && j < N)
		Output[index] = A[index] + C[index];
}


int main(int argc, char** argv) {
	hipError_t cudaStat1, cudaStat2, cudaStat3;
	hipsparseHandle_t handle = 0;
	hipsparseStatus_t status;
	hipsparseMatDescr_t descr = 0;
	FILE* fp;
	int rows = 0, curr_row = 0, curr_col;
	char *token, filename[50], c, str[50], line[50]= "";
	size_t len = 20;
	char* dilimeter = ",";
	printf("Enter data file name: ");
	scanf("%s", filename);
	printf("\n");
	//READ FILE//
	//Count file lines
	fp = fopen(filename, "r");
	if (fp == NULL) {
		printf("Error on reading file.");
		return 0;
	}
	for (c = getc(fp); c != EOF; c = getc(fp))
		if (c == '\n')	rows = rows + 1;
	rewind(fp);
	printf("The file %s has %d lines\n ", filename, rows);
	//Create Array
	int nnz = rows,count,n,*z=0;
	int* cooRowIndexHostPtr = 0;
	int* cooColIndexHostPtr = 0;
	int* cooValHostPtr = 0;
	int* cooRowIndex = 0;
	int* cooColIndex = 0;
	int* cooVal = 0;
	int* csrRowPtr = 0;
	cooRowIndexHostPtr = (int*)malloc(nnz * sizeof(cooRowIndexHostPtr[0]));
	cooColIndexHostPtr = (int*)malloc(nnz * sizeof(cooColIndexHostPtr[0]));
	cooValHostPtr = (int*)malloc(nnz * sizeof(cooValHostPtr[0]));
		//
		n = 448695;//Array dimension
	//Read data line by line (data is in COO format exported by Matlab)
	while (fgets(line, sizeof(line), fp)) {
		//printf("%s",line);
		token = strtok(line, dilimeter);
		count = 0;
		while (token != NULL) {
			if (count == 0)
				cooRowIndexHostPtr[curr_row] = (int)atof(token);//string->float->int
			else
				cooColIndexHostPtr[curr_row] = (int)atof(token);
			token = strtok(NULL, dilimeter);
			count++;
		}
		cooValHostPtr[curr_row] = 1;
		curr_row++;
	}
	/*
	for (int i = 1;i < 20;i++) {
		printf("%d %d %d\n", cooRowIndexHostPtr[i], cooColIndexHostPtr[i], cooValHostPtr[i]);
	}
		printf("%d %d %d\n", cooRowIndexHostPtr[nnz - 1], cooColIndexHostPtr[nnz - 1], cooValHostPtr[nnz - 1]);
	*/
	//Allocate GPU Memory and copy data
	cudaStat1 = hipMalloc((void**)&cooRowIndex, nnz * sizeof(cooRowIndex[0]));
	cudaStat2 = hipMalloc((void**)&cooColIndex, nnz * sizeof(cooColIndex[0]));
	cudaStat3 = hipMalloc((void**)&cooVal, nnz * sizeof(cooVal[0]));
	if ((cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess)) {
		printf("Device Malloc failed");
		return 1;
	}
	cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, (size_t)(nnz*sizeof(cooRowIndex[0])), hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr, (size_t)(nnz*sizeof(cooColIndex[0])), hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(cooVal[0])), hipMemcpyHostToDevice);
	if ((cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess)) {
		printf("Device Malloc failed");
		return 1;
	}
	//Init cusparse
	status = hipsparseCreate(&handle);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Error: CUSPARSE library initialization failed.");
		return 1;
	}
	status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Error: CUSPARSE matrix descriptor initialization failed.");
		return 1;
	}
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	//Convert COO to CSR format
	cudaStat1 = hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(csrRowPtr[0]));
	status = hipsparseXcoo2csr(handle, cooRowIndex, nnz, n, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Error: Conversion from COO to CSR format failed.");
		return 1;
	}
	//
	int devId;
	hipDeviceProp_t prop;
	hipError_t cudaStat;
	cudaStat = hipGetDevice(&devId);
	if (hipSuccess != cudaStat) {
		printf("hipGetDevice failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}
	cudaStat = hipGetDeviceProperties(&prop, devId);
	if (hipSuccess != cudaStat) {
		printf("hipGetDeviceProperties failed");
		printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
		return 1;
	}
	//FIND C=A*A
	int baseC, nnzC;
	int* csrRowPtrC=0;
	int* csrValC=0;
	int* csrColIndC=0;
	int *nnzTotalDevHostPtr = &nnzC;
	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
	hipMalloc((void**)&csrRowPtrC, sizeof(csrRowPtrC[0]) * (n + 1));
	hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, n, descr, nnz,
		csrRowPtr, cooColIndex, descr, nnz, csrRowPtr, cooColIndex, descr, csrRowPtrC, nnzTotalDevHostPtr);
	//To nnzTotalDevHostPtr (αριθμός μη-μηδενικών του γινομένου) βγαίνει λάθος για κάποιο λόγο, ίσως κάποιο overflow ή segmentation fault που δε μπορώ να βρώ
	if (NULL != nnzTotalDevHostPtr) {
		nnzC = *nnzTotalDevHostPtr;
		printf("1\n");
	}
	else {
		hipMemcpy(&nnzC, csrRowPtrC + n, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
		nnzC -= baseC;
		printf("2\n");
	}
	nnzC = 33050793;//Το βρήκα από Matlab, έβγαινε λάθος εδώ αλλά ο κώδικας υπάρχει παραπάνω
	printf("%d\n", nnzC);
	hipMalloc((void**)&csrColIndC, sizeof(int) * nnzC);
	hipMalloc((void**)&csrValC, sizeof(int) * nnzC);
	status = hipsparseDcsrgemm(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,n,n,n,descr,nnz,
		(double*)cooVal,csrRowPtr,cooColIndex,descr,nnz,(double*)cooVal,csrRowPtr,cooColIndex,
		descr,(double*)csrValC,csrRowPtrC,csrColIndC);
	printf("First product complete.\n");
	//WORKS UNTIL HERE, THE PART BELOW IS NOT COMPLETE
	/*
	int* CSR_Host_RowIndexHostPtr=0;
	int* CSR_Host_ColIndexHostPtr=0;
	int* CSR_Host_ValHostPtr=0;
	CSR_Host_RowIndexHostPtr = (int*)malloc((n+1) * sizeof(CSR_Host_RowIndexHostPtr[0]));
	CSR_Host_ColIndexHostPtr = (int*)malloc(nnzC * sizeof(CSR_Host_ColIndexHostPtr[0]));
	CSR_Host_ValHostPtr = (int*)malloc(nnzC * sizeof(CSR_Host_ValHostPtr[0]));
	hipMemcpy(&CSR_Host_RowIndexHostPtr, csrRowPtrC, (size_t)(n + 1) * CSR_Host_RowIndexHostPtr[0], hipMemcpyDeviceToHost);
	hipMemcpy(&CSR_Host_ColIndexHostPtr, csrColIndC, (size_t)(nnzC) * CSR_Host_ColIndexHostPtr[0], hipMemcpyDeviceToHost);
	hipMemcpy(&CSR_Host_ValHostPtr, csrValC, (size_t)(nnzC) * CSR_Host_ValHostPtr[0], hipMemcpyDeviceToHost);
	*/
	int* tempRowC = 0;
	int* tempValC = 0;
	int* tempColC = 0;
	int* tempRowA = 0;
	int* tempValA = 0;
	int* tempColA = 0;
	hipMalloc((void**)&csrColIndC, sizeof(int) * nnzC);
	//init kernel
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(n / dimBlock.x, n / dimBlock.y);
	int* A = 0;
	int* C = 0;
	int* Out = 0;
	hipMalloc((void**)&C, sizeof(int) * (128*n));//temp array C
	hipMalloc((void**)&A, sizeof(int)* (128 * n));//temp array A
	hipMalloc((void**)&Out, sizeof(int)* (128 * n));//temp array A
	//Scatter blocks of the two arrays and do Hadamard multiplication 128 rows at a time
	for (int k = 0;k < n/128;k++) {
		hipsparseDcsr2dense(handle, 128, n, descr,(double*)csrValC,csrRowPtrC+k,csrColIndC,(double*)C,128 );
		hipsparseDcsr2dense(handle, 128, n, descr,(double*)cooVal,csrRowPtr+k,cooColIndex,(double*)A,128 );
		Hadamard_kernel <<<dimGrid, dimBlock >>> (A,C,Out,n);
	}
	//results
	printf("\nTime taken: %d");
}